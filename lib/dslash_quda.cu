#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <color_spinor_field.h>

#define BLOCK_DIM 64

// these control the Wilson-type actions
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR

//these are access control for staggered action
#if (__CUDA_ARCH__ >= 200)
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
#else
#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
#endif

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>

enum KernelType {
  INTERIOR_KERNEL = 5,
  EXTERIOR_KERNEL_X = 0,
  EXTERIOR_KERNEL_Y = 1,
  EXTERIOR_KERNEL_Z = 2,
  EXTERIOR_KERNEL_T = 3
};

struct DslashParam {
  int tOffset; // offset into the T dimension (multi gpu only)
  int tMul;    // spatial volume distance between the T faces being updated (multi gpu only)
  int threads; // the desired number of active threads
  int parity;  // Even-Odd or Odd-Even
  int commDim[QUDA_MAX_DIM]; // Whether to do comms or not
  int ghostDim[QUDA_MAX_DIM]; // Whether a ghost zone has been allocated for a given dimension
  int ghostOffset[QUDA_MAX_DIM];
  int ghostNormOffset[QUDA_MAX_DIM];
  KernelType kernel_type; //is it INTERIOR_KERNEL, EXTERIOR_KERNEL_X/Y/Z/T
};

// determines whether the temporal ghost zones are packed with a gather kernel,
// as opposed to multiple calls to hipMemcpy()
//! WARNING:for DW must be currently set to 'true'
bool kernelPackT = false; 

DslashParam dslashParam;

// these are set in initDslashConst
int Vspatial;
#ifdef MULTI_GPU
static const int Nstream = 9;
#else
static const int Nstream = 1;
#endif
static hipStream_t streams[Nstream];

FaceBuffer *face;
cudaColorSpinorField *inSpinor;

#include <dslash_textures.h>
#include <dslash_constants.h>

#define SHORT_LENGTH 65536
#define SCALE_FLOAT ((SHORT_LENGTH-1) * 0.5) // 32767.5
#define SHIFT_FLOAT (-1.f / (SHORT_LENGTH-1)) // 1.5259021897e-5

#if defined(DIRECT_ACCESS_LINK) || defined(DIRECT_ACCESS_WILSON_SPINOR) || \
  defined(DIRECT_ACCESS_WILSON_ACCUM) || defined(DIRECT_ACCESS_WILSON_PACK_SPINOR)
static inline __device__ short float2short(float c, float a) {
  //return (short)(a*MAX_SHORT);
  short rtn = (short)((a+SHIFT_FLOAT)*SCALE_FLOAT*c);
  return rtn;
}

static inline __device__ float short2float(short a) {
  return (float)a/SCALE_FLOAT - SHIFT_FLOAT;
}

static inline __device__ short4 float42short4(float c, float4 a) {
  return make_short4(float2short(c, a.x), float2short(c, a.y), float2short(c, a.z), float2short(c, a.w));
}

static inline __device__ float4 short42float4(short4 a) {
  return make_float4(short2float(a.x), short2float(a.y), short2float(a.z), short2float(a.w));
}

static inline __device__ float2 short22float2(short2 a) {
  return make_float2(short2float(a.x), short2float(a.y));
}
#endif // DIRECT_ACCESS inclusions

#include <pack_face_def.h>        // kernels for packing the ghost zones and general indexing
#include <staggered_dslash_def.h> // staggered Dslash kernels
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#include <dw_dslash_def.h>        // Domain Wall kernels
#include <tm_dslash_def.h>        // Twisted Mass kernels
#include <tm_core.h>              // solo twisted mass kernel
#include <clover_def.h>           // kernels for applying the clover term alone

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef CLOVER_SHARED_FLOATS_PER_THREAD
#define CLOVER_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef SHARED_COORDS
#define SHARED_COORDS 0
#endif

#include <blas_quda.h>
#include <face_quda.h>


// dslashTuning = QUDA_TUNE_YES turns off error checking
static QudaTune dslashTuning = QUDA_TUNE_NO;

void setDslashTuning(QudaTune tune)
{
  dslashTuning = tune;
}

__global__ void dummyKernel() {
  // do nothing
}

void initCache() {

#if (__CUDA_ARCH__ >= 200)

  static int firsttime = 1;
  if (firsttime){	
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dummyKernel), hipFuncCachePreferL1);
    dummyKernel<<<1,1>>>();
    firsttime=0;
  }

#endif

}

void setFace(const FaceBuffer &Face) {
  face = (FaceBuffer*)&Face; // nasty
}

#ifndef MULTI_GPU

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
 case INTERIOR_KERNEL:							\
    if (x==0) {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
      } else {								\
	FUNC ## 8 ## DAG ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    } else {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## X ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## X ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
	FUNC ## 8 ## DAG ## X ## Kernel<INTERIOR_KERNEL> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    }									\
    break;								\
  default:								\
   errorQuda("KernelType %d not defined for single GPU", param.kernel_type); \
 }

#else

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
 case INTERIOR_KERNEL:							\
    if (x==0) {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
      } else {								\
	FUNC ## 8 ## DAG ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    } else {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## X ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## X ## Kernel<INTERIOR_KERNEL><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
	FUNC ## 8 ## DAG ## X ## Kernel<INTERIOR_KERNEL> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    }									\
    break;\
 case EXTERIOR_KERNEL_X:						\
    if (x==0) {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## Kernel<EXTERIOR_KERNEL_X><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## Kernel<EXTERIOR_KERNEL_X><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else {								\
	FUNC ## 8 ## DAG ## Kernel<EXTERIOR_KERNEL_X><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    } else {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_X><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_X><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
	FUNC ## 8 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_X><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    }									\
 break;									\
 case EXTERIOR_KERNEL_Y:						\
    if (x==0) {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## Kernel<EXTERIOR_KERNEL_Y><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## Kernel<EXTERIOR_KERNEL_Y><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else {								\
	FUNC ## 8 ## DAG ## Kernel<EXTERIOR_KERNEL_Y><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    } else {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_Y><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_Y><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
	FUNC ## 8 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_Y><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    }									\
    break;								\
 case EXTERIOR_KERNEL_Z:						\
    if (x==0) {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## Kernel<EXTERIOR_KERNEL_Z><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## Kernel<EXTERIOR_KERNEL_Z><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else {								\
	FUNC ## 8 ## DAG ## Kernel <EXTERIOR_KERNEL_Z><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    } else {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_Z><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_Z><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
	FUNC ## 8 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_Z><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    }									\
    break;								\
 case EXTERIOR_KERNEL_T:						\
    if (x==0) {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## Kernel<EXTERIOR_KERNEL_T><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## Kernel<EXTERIOR_KERNEL_T><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else {								\
	FUNC ## 8 ## DAG ## Kernel<EXTERIOR_KERNEL_T><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    } else {								\
      if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
	FUNC ## 18 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_T><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
	FUNC ## 12 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_T><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
	FUNC ## 8 ## DAG ## X ## Kernel<EXTERIOR_KERNEL_T><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
      }									\
    }									\
    break;								\
  }

#endif

// macro used for dslash types with dagger kernel defined (Wilson, domain wall, etc.)
#define DSLASH(FUNC, gridDim, blockDim, shared, stream, param, ...)	\
  if (!dagger) {							\
    GENERIC_DSLASH(FUNC, , Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
  } else {								\
    GENERIC_DSLASH(FUNC, Dagger, Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
 }

// macro used for staggered dslash
#define STAGGERED_DSLASH(gridDim, blockDim, shared, stream, param, ...)	\
    GENERIC_DSLASH(staggeredDslash, , Axpy, gridDim, blockDim, shared, stream, param, __VA_ARGS__)


// Use an abstract class interface to drive the different CUDA dslash
// kernels.  All parameters are curried into the derived classes to
// allow a simple interface.
class DslashCuda {
public:
  DslashCuda() { ; }
  virtual ~DslashCuda() { ; }
  virtual void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) = 0;
};


template <typename sFloat, typename gFloat>
class WilsonDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  const double a;

public:
  WilsonDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		   const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		   const sFloat *x, const float *xNorm, const double a,
		   const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), in(in), 
    inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
  }

  virtual ~WilsonDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(dslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
	   out, outNorm, gauge0, gauge1, in, inNorm, x, xNorm, a);
  }

};

template <typename sFloat, typename gFloat, typename cFloat>
class CloverDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const cFloat *clover;
  const float *cloverNorm;
  const int dagger;
  const double a;

public:
  CloverDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		   const QudaReconstructType reconstruct, const cFloat *clover, 
		   const float *cloverNorm, const sFloat *in, const float *inNorm,
		   const sFloat *x, const float *xNorm, const double a,
		   const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    clover(clover), cloverNorm(cloverNorm), in(in), inNorm(inNorm), 
    reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
  }
  virtual ~CloverDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(cloverDslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
	   out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, x, xNorm, a);
  }

};

void setTwistParam(double &a, double &b, const double &kappa, const double &mu, 
		   const int dagger, const QudaTwistGamma5Type twist) {
  if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
    a = 2.0 * kappa * mu;
    b = 1.0;
  } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
    a = -2.0 * kappa * mu;
    b = 1.0 / (1.0 + a*a);
  } else {
    errorQuda("Twist type %d not defined\n", twist);
  }
  if (dagger) a *= -1.0;

}

template <typename sFloat, typename gFloat>
class TwistedDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  double a;
  double b;

public:
  TwistedDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		    const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		    const sFloat *x, const float *xNorm, const double kappa, const double mu,
		    const double k, const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    in(in), inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
    setTwistParam(a, b, kappa, mu, dagger, QUDA_TWIST_GAMMA5_INVERSE);
    if (x) b *= k;
  }
  virtual ~TwistedDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(twistedMassDslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
	   out, outNorm, gauge0, gauge1, in, inNorm, a, b, x, xNorm);
  }

};

template <typename sFloat, typename gFloat>
class DomainWallDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  const double mferm;
  const double a;

public:
  DomainWallDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		       const QudaReconstructType reconstruct, const sFloat *in, 
		       const float *inNorm, const sFloat *x, const float *xNorm, const double mferm, 
		       const double a, const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    in(in), inNorm(inNorm), mferm(mferm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
  }
  virtual ~DomainWallDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(domainWallDslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
    	   out, outNorm, gauge0, gauge1, in, inNorm, mferm, x, xNorm, a);
  }

};

void dslashCuda(DslashCuda &dslash, const size_t regSize, const int parity, const int dagger, 
		const int volume, const int *faceVolumeCB, const dim3 *blockDim) {

  dslashParam.parity = parity;

  dslashParam.kernel_type = INTERIOR_KERNEL;

  dslashParam.tOffset = 0;//! 
  dslashParam.tMul = 1;   //! 
  dslashParam.threads = volume;

#ifdef MULTI_GPU
    // Gather from source spinor
  for(int dir = 3; dir >=0; dir--){ // count down for Wilson
    if (!dslashParam.commDim[dir]) continue;
    face->exchangeFacesStart(*inSpinor, 1-parity, dagger, dir, streams);
  }
#endif

  int shared_bytes = blockDim[0].x*(DSLASH_SHARED_FLOATS_PER_THREAD*regSize + SHARED_COORDS);
  dslash.apply(blockDim[0], shared_bytes, streams[Nstream-1]); // stream 0 or 8

#ifdef MULTI_GPU

  for (int i=3; i>=0; i--) { // count down for Wilson
    if (!dslashParam.commDim[i]) continue;

    // Finish gather and start comms
    face->exchangeFacesComms(i);
    
    // Wait for comms to finish, and scatter into the end zone
    face->exchangeFacesWait(*inSpinor, dagger, i);
  }

  for (int i=3; i>=0; i--) { // count down for Wilson
    if (!dslashParam.commDim[i]) continue;

    shared_bytes = blockDim[i+1].x*(DSLASH_SHARED_FLOATS_PER_THREAD*regSize + SHARED_COORDS);
    
    hipStreamSynchronize(streams[2*i]);
    hipStreamSynchronize(streams[2*i + 1]);
    dslashParam.kernel_type = static_cast<KernelType>(i);
    //dslashParam.tOffset = dims[i]-2; // is this redundant?
    dslashParam.threads = 2*faceVolumeCB[i]; // updating 2 faces

    dslash.apply(blockDim[i+1], shared_bytes, streams[Nstream-1]); // all faces use this stream
  }
  hipStreamSynchronize(streams[Nstream-1]);
#endif // MULTI_GPU
}

// Wilson wrappers
void wilsonDslashCuda(cudaColorSpinorField *out, const FullGauge gauge, const cudaColorSpinorField *in,
		      const int parity, const int dagger, const cudaColorSpinorField *x,
		      const double &k, const dim3 *blockDim, const int *commOverride) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_WILSON_DIRAC
  int Npad = (in->nColor*in->nSpin*2)/in->fieldOrder; // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->ghostOffset[i] + in->stride);
    dslashParam.ghostNormOffset[i] = in->ghostNormOffset[i] + in->stride;
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  void *xv = (x ? x->v : 0);
  void *xn = (x ? x->norm : 0);

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);
  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new WilsonDslashCuda<double2, double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, (double2*)gauge1, 
						    gauge.reconstruct, (double2*)in->v, (float*)in->norm, 
						    (double2*)xv, (float*)xn, k, dagger, in->bytes, in->norm_bytes);
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new WilsonDslashCuda<float4, float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, (float4*)gauge1,
						  gauge.reconstruct, (float4*)in->v, (float*)in->norm, 
						  (float4*)xv, (float*)xn, k, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new WilsonDslashCuda<short4, short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, (short4*)gauge1,
						  gauge.reconstruct, (short4*)in->v, (float*)in->norm,
						  (short4*)xv, (float*)xn, k, dagger, in->bytes, in->norm_bytes);
  }
  dslashCuda(*dslash, regSize, parity, dagger, in->volume, in->ghostFace, blockDim);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC

}

void cloverDslashCuda(cudaColorSpinorField *out, const FullGauge gauge, const FullClover cloverInv,
		      const cudaColorSpinorField *in, const int parity, const int dagger, 
		      const cudaColorSpinorField *x, const double &a,
		      const dim3 *blockDim, const int *commOverride) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_CLOVER_DIRAC
  int Npad = (in->nColor*in->nSpin*2)/in->fieldOrder; // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->ghostOffset[i] + in->stride);
    dslashParam.ghostNormOffset[i] = in->ghostNormOffset[i] + in->stride;
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }

  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (in->precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  void *xv = x ? x->v : 0;
  void *xn = x ? x->norm : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new CloverDslashCuda<double2, double2, double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, 
							     (double2*)gauge1, gauge.reconstruct, (double2*)cloverP, 
							     (float*)cloverNormP, (double2*)in->v, (float*)in->norm,
							     (double2*)xv, (float*)xn, a, dagger, in->bytes, in->norm_bytes);
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new CloverDslashCuda<float4, float4, float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, 
							  (float4*)gauge1, gauge.reconstruct, (float4*)cloverP, 
							  (float*)cloverNormP, (float4*)in->v, (float*)in->norm, 
							  (float4*)xv, (float*)xn, a, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new CloverDslashCuda<short4, short4, short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, 
							  (short4*)gauge1, gauge.reconstruct, (short4*)cloverP, 
							  (float*)cloverNormP, (short4*)in->v, (float*)in->norm, 
							  (short4*)xv, (float*)xn, a, dagger, in->bytes, in->norm_bytes);
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->volume, in->ghostFace, blockDim);

  delete dslash;
  unbindGaugeTex(gauge);
  unbindCloverTex(cloverInv);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}


void twistedMassDslashCuda(cudaColorSpinorField *out, const FullGauge gauge, 
			   const cudaColorSpinorField *in, const int parity, const int dagger, 
			   const cudaColorSpinorField *x, const double &kappa, const double &mu, 
			   const double &a, const dim3 *blockDim, const int *commOverride) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_TWISTED_MASS_DIRAC
  int Npad = (in->nColor*in->nSpin*2)/in->fieldOrder; // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->ghostOffset[i] + in->stride);
    dslashParam.ghostNormOffset[i] = in->ghostNormOffset[i] + in->stride;
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  void *xv = x ? x->v : 0;
  void *xn = x ? x->norm : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new TwistedDslashCuda<double2,double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, 
						    (double2*)gauge1, gauge.reconstruct, (double2*)in->v, 
						    (float*)in->norm, (double2*)xv, (float*)xn, 
						    kappa, mu, a, dagger, in->bytes, in->norm_bytes);
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new TwistedDslashCuda<float4,float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, (float4*)gauge1, 
						  gauge.reconstruct, (float4*)in->v, (float*)in->norm, 
						  (float4*)xv, (float*)xn, kappa, mu, a, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new TwistedDslashCuda<short4,short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, (short4*)gauge1, 
						  gauge.reconstruct, (short4*)in->v, (float*)in->norm, 
						  (short4*)xv, (float*)xn, kappa, mu, a, dagger, in->bytes, in->norm_bytes);
    
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->volume, in->ghostFace, blockDim);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif

}

void domainWallDslashCuda(cudaColorSpinorField *out, const FullGauge gauge, 
			  const cudaColorSpinorField *in, const int parity, const int dagger, 
			  const cudaColorSpinorField *x, const double &m_f, const double &k2,
			  const dim3 *blockDim, const int *commOverride) {//new arg added

  inSpinor = (cudaColorSpinorField*)in; // EVIL

//#ifdef MULTI_GPU
//  errorQuda("Multi-GPU domain wall not implemented\n");
//#endif

  dslashParam.parity = parity;
  dslashParam.threads = in->volume;//!check this: must be 5d cb volume

#ifdef GPU_DOMAIN_WALL_DIRAC
//BEGIN NEW
  kernelPackT = true; 
  
  int Npad = (in->nColor*in->nSpin*2)/in->fieldOrder; // SPINOR_HOP in old code
  
  //currently splitting in space-time is impelemented:
  //!check for 'in' object: ghostOffset, ghostNormOffset and ghostFace data elements
  int dirs = 4;
  for(int i = 0;i < dirs; i++)
  {
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->ghostOffset[i] + in->stride);
    dslashParam.ghostNormOffset[i] = in->ghostNormOffset[i] + in->stride;
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    //printf("%d ghostDim = %d commDim = %d\n", i, dslashParam.ghostDim[i], dslashParam.commDim[i]);
  }
//END NEW

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  void *xv = (x ? x->v : 0);
  void *xn = (x ? x->norm : 0);

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);


  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new DomainWallDslashCuda<double2,double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, (double2*)gauge1, 
						       gauge.reconstruct, (double2*)in->v, (float*)in->norm, (double2*)xv, 
						       (float*)xn, m_f, k2, dagger, in->bytes, in->norm_bytes);
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new DomainWallDslashCuda<float4,float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, (float4*)gauge1, 
						     gauge.reconstruct, (float4*)in->v, (float*)in->norm, (float4*)xv, 
						     (float*)xn, m_f, k2, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new DomainWallDslashCuda<short4,short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, (short4*)gauge1, 
						     gauge.reconstruct, (short4*)in->v, (float*)in->norm, (short4*)xv, 
						     (float*)xn, m_f, k2, dagger, in->bytes, in->norm_bytes);
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->volume, in->ghostFace, blockDim);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Domain wall dslash has not been built");
#endif

}


template <typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
  void staggeredDslashCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
			   const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
			   const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
			   const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
			   const double &a, const int volume, const int* Vsh, const int* dims,
			   const int length, const int ghost_length, const dim3 *blockDim) {
    
  dim3 interiorGridDim( (dslashParam.threads + blockDim[0].x -1)/blockDim[0].x, 1, 1);
  dim3 exteriorGridDim[4]  = {
    dim3((6*Vsh[0] + blockDim[1].x -1)/blockDim[1].x, 1, 1),
    dim3((6*Vsh[1] + blockDim[2].x -1)/blockDim[2].x, 1, 1),
    dim3((6*Vsh[2] + blockDim[3].x -1)/blockDim[3].x, 1, 1),
    dim3((6*Vsh[3] + blockDim[4].x -1)/blockDim[4].x, 1, 1)
  };
    
  size_t regSize = bindSpinorTex_mg(length, ghost_length, in, inNorm, x, xNorm); CUERR;
  int shared_bytes = blockDim[0].x*6*regSize;
  
  dslashParam.kernel_type = INTERIOR_KERNEL;
  dslashParam.tOffset =  0;
  dslashParam.threads = volume;
#ifdef MULTI_GPU
  // Gather from source spinor
  for(int dir = 0; dir <4; dir++){
    if (!dslashParam.commDim[dir]) continue;
    face->exchangeFacesStart(*inSpinor, 1-parity, dagger, dir, streams);
  }
#endif

  STAGGERED_DSLASH(interiorGridDim, blockDim[0], shared_bytes, streams[Nstream-1], dslashParam,
		   out, outNorm, fatGauge0, fatGauge1, longGauge0, longGauge1, in, inNorm, x, xNorm, a); CUERR;

#ifdef MULTI_GPU

  for(int i=0 ;i < 4;i++){
    if (!dslashParam.commDim[i]) continue;

    // Finish gather and start comms
    face->exchangeFacesComms(i);
    // Wait for comms to finish, and scatter into the end zone
    face->exchangeFacesWait(*inSpinor, dagger,i);    
  }

  for(int i=0 ;i < 4;i++){
    if(!dslashParam.commDim[i]) continue;

    shared_bytes = blockDim[i+1].x*6*regSize;
    
    hipStreamSynchronize(streams[2*i]);
    hipStreamSynchronize(streams[2*i + 1]);
    dslashParam.kernel_type = static_cast<KernelType>(i);
    dslashParam.tOffset =  dims[i]-6;
    dslashParam.threads = 6*Vsh[i];
    STAGGERED_DSLASH(exteriorGridDim[i], blockDim[i+1], shared_bytes, streams[Nstream-1], dslashParam,
		     out, outNorm, fatGauge0, fatGauge1, longGauge0, longGauge1, in, inNorm, x, xNorm, a); CUERR;
  }
  hipStreamSynchronize(streams[Nstream-1]);

#endif
}

void staggeredDslashCuda(cudaColorSpinorField *out, const FullGauge fatGauge, 
			 const FullGauge longGauge, const cudaColorSpinorField *in,
			 const int parity, const int dagger, const cudaColorSpinorField *x,
			 const double &k, const dim3 *block, const int *commOverride)
{
  
  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_STAGGERED_DIRAC

  dslashParam.parity = parity;
  dslashParam.threads = in->volume;
  int Npad = (in->nColor*in->nSpin*2)/in->fieldOrder; // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->ghostOffset[i] + in->stride);
    dslashParam.ghostNormOffset[i] = in->ghostNormOffset[i] + in->stride;
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }
  void *fatGauge0, *fatGauge1;
  void* longGauge0, *longGauge1;
  bindFatGaugeTex(fatGauge, parity, &fatGauge0, &fatGauge1);
  bindLongGaugeTex(longGauge, parity, &longGauge0, &longGauge1);
    
  if (in->precision != fatGauge.precision || in->precision != longGauge.precision){
    errorQuda("Mixing gauge and spinor precision not supported"
	      "(precision=%d, fatlinkGauge.precision=%d, longGauge.precision=%d",
	      in->precision, fatGauge.precision, longGauge.precision);
  }
    
  void *xv = x ? x->v : 0;
  void *xn = x ? x->norm : 0;

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    staggeredDslashCuda((double2*)out->v, (float*)out->norm, (double2*)fatGauge0, (double2*)fatGauge1,
			(double2*)longGauge0, (double2*)longGauge1, longGauge.reconstruct, 
			(double2*)in->v, (float*)in->norm, parity, dagger, 
			(double2*)xv, (float*)x, k, in->volume, in->ghostFace, 
			in->x, in->length, in->ghost_length, block);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    staggeredDslashCuda((float2*)out->v, (float*)out->norm, (float2*)fatGauge0, (float2*)fatGauge1,
			(float4*)longGauge0, (float4*)longGauge1, longGauge.reconstruct, 
			(float2*)in->v, (float*)in->norm, parity, dagger, 
			(float2*)xv, (float*)xn, k, in->volume, in->ghostFace, 
			in->x, in->length, in->ghost_length, block);
  } else if (in->precision == QUDA_HALF_PRECISION) {	
    staggeredDslashCuda((short2*)out->v, (float*)out->norm, (short2*)fatGauge0, (short2*)fatGauge1,
			(short4*)longGauge0, (short4*)longGauge1, longGauge.reconstruct, 
			(short2*)in->v, (float*)in->norm, parity, dagger, 
			(short2*)xv, (float*)xn, k, in->volume, in->ghostFace, 
			in->x, in->length, in->ghost_length, block);
  }

  if (!dslashTuning) checkCudaError();
  
#else
  errorQuda("Staggered dslash has not been built");
#endif  
}


template <typename spinorFloat, typename cloverFloat>
void cloverCuda(spinorFloat *out, float *outNorm, const cloverFloat *clover,
		const float *cloverNorm, const spinorFloat *in, const float *inNorm, 
		const size_t bytes, const size_t norm_bytes, const dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  int shared_bytes = blockDim.x*CLOVER_SHARED_FLOATS_PER_THREAD*bindSpinorTex(bytes, norm_bytes, in, inNorm);
  cloverKernel<<<gridDim, blockDim, shared_bytes>>> 
    (out, outNorm, clover, cloverNorm, in, inNorm, dslashParam);
  unbindSpinorTex(in, inNorm);
}

void cloverCuda(cudaColorSpinorField *out, const FullGauge gauge, const FullClover clover, 
		const cudaColorSpinorField *in, const int parity, const dim3 &blockDim) {

  dslashParam.parity = parity;
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = in->volume;

#ifdef GPU_CLOVER_DIRAC
  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(clover, parity, &cloverP, &cloverNormP);

  if (in->precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    cloverCuda((double2*)out->v, (float*)out->norm, (double2*)cloverP, 
	       (float*)cloverNormP, (double2*)in->v, (float*)in->norm, 
	       in->bytes, in->norm_bytes, blockDim);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    cloverCuda((float4*)out->v, (float*)out->norm, (float4*)cloverP, 
	       (float*)cloverNormP, (float4*)in->v, (float*)in->norm,
	       in->bytes, in->norm_bytes, blockDim);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    cloverCuda((short4*)out->v, (float*)out->norm, (short4*)cloverP, 
	       (float*)cloverNormP, (short4*)in->v, (float*)in->norm, 
	       in->bytes, in->norm_bytes, blockDim);
  }
  unbindCloverTex(clover);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}
// FIXME: twist kernel cannot be issued asynchronously because of texture unbinding
template <typename spinorFloat>
void twistGamma5Cuda(spinorFloat *out, float *outNorm, const spinorFloat *in, 
		     const float *inNorm, const int dagger, const double &kappa, 
		     const double &mu, const size_t bytes, const size_t norm_bytes, 
		     const QudaTwistGamma5Type twist, dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  double a=0.0, b=0.0;
  setTwistParam(a, b, kappa, mu, dagger, twist);

  bindSpinorTex(bytes, norm_bytes, in, inNorm);
  twistGamma5Kernel<<<gridDim, blockDim, 0>>> (out, outNorm, a, b, dslashParam);
  unbindSpinorTex(in, inNorm);
}

void twistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		     const int dagger, const double &kappa, const double &mu,
		     const QudaTwistGamma5Type twist, const dim3 &block) {

  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = in->Volume();

#ifdef GPU_TWISTED_MASS_DIRAC
  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    twistGamma5Cuda((double2*)out->v, (float*)out->norm, 
		    (double2*)in->v, (float*)in->norm, 
		    dagger, kappa, mu, in->bytes, 
		    in->norm_bytes, twist, block);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    twistGamma5Cuda((float4*)out->v, (float*)out->norm,
		    (float4*)in->v, (float*)in->norm, 
		    dagger, kappa, mu, in->bytes, 
		    in->norm_bytes, twist, block);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    twistGamma5Cuda((short4*)out->v, (float*)out->norm,
		    (short4*)in->v, (float*)in->norm, 
		    dagger, kappa, mu, in->bytes, 
		    in->norm_bytes, twist, block);
  }
  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
}


#include "misc_helpers.cu"


#if defined(GPU_FATLINK)||defined(GPU_GAUGE_FORCE)|| defined(GPU_FERMION_FORCE)
#include <force_common.h>
#include "force_kernel_common.cu"
#endif

#ifdef GPU_FATLINK
#include "llfat_quda.cu"
#endif

#ifdef GPU_GAUGE_FORCE
#include "gauge_force_quda.cu"
#endif

#ifdef GPU_FERMION_FORCE
#include "fermion_force_quda.cu"
#endif
